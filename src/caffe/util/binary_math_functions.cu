#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
#ifndef CPU_ONLY

#define KERNEL_FUNC(NAME, OP, PARAMS...) \
  template <typename Dtype> \
  __global__ void NAME(const int M, const int N, PARAMS)\
  { \
    int i = blockIdx.x * blockDim.x + threadIdx.x; \
    int j = blockIdx.y * blockDim.y + threadIdx.y; \
    if (i < M && j < N) { \
      OP; \
    } \
  }

KERNEL_FUNC(
  binary_gradient_kernel_0,
  grad[i * N + j] *=
    mul + (fabs(in[i * N + j]) <= Dtype(1) ? scale[i] : Dtype(0)),
  const Dtype *in, const Dtype *scale, Dtype *grad, const Dtype mul);
KERNEL_FUNC(
  binary_gradient_kernel_1,
  grad[i * N + j] *=
    mul + (fabs(in[i * N + j]) <= Dtype(1) ? scale[j] : Dtype(0)),
  const Dtype *in, const Dtype *scale, Dtype *grad, const Dtype mul);

template<typename Dtype>
void caffe_gpu_binary_gradient(
  const int axis, const int M, const int N,
  const Dtype *in, const Dtype *scale, Dtype *grad) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    const Dtype mul = 1. / N;
    binary_gradient_kernel_0<Dtype> <<<blocks, threads>>> (
      M, N, in, scale, grad, mul);
  }
  else {
    const Dtype mul = 1. / M;
    binary_gradient_kernel_1<Dtype> <<<blocks, threads>>> (
      M, N, in, scale, grad, mul);
  }
}

template<typename Dtype>
__global__ void ternary_gradient_kernel_0(
  const int M, const int N, const Dtype *in, const Dtype *scale,
  const Dtype *delta, Dtype *grad, const Dtype mul) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < M && y < N) {
    register Dtype val = fabs(in[x * N + y]);
    grad[x * N + y] *= mul + Dtype(val <= delta[x] ? 1 : val > Dtype(1) ? 0 :
                                   scale[x]);
  }
}

template<typename Dtype>
__global__ void ternary_gradient_kernel_1(
  const int M, const int N, const Dtype *in, const Dtype *scale,
  const Dtype *delta, Dtype *grad, const Dtype mul) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < M && y < N) {
    register Dtype val = fabs(in[x * N + y]);
    grad[x * N + y] *= mul + Dtype(val <= delta[y] ? 1 : val > Dtype(1) ? 0 :
                                   scale[y]);
  }
}

template<typename Dtype>
void caffe_gpu_ternary_gradient(
  const int axis, const int M, const int N,
  const Dtype *in, const Dtype *scale, const Dtype *delta, Dtype *grad) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    const Dtype mul = 1. / N;
    ternary_gradient_kernel_0<Dtype> <<<blocks, threads>>> (
      M, N, in, scale, delta, grad, mul);
  }
  else {
    const Dtype mul = 1. / M;
    ternary_gradient_kernel_1<Dtype> <<<blocks, threads>>> (
      M, N, in, scale, delta, grad, mul);
  }
}

template <typename Dtype>
__global__ void clip_kernel(
  const int N, const Dtype min_value, const Dtype max_value, Dtype *X);
template <>
__global__ void clip_kernel<float>(
  const int N, const float min_value, const float max_value, float *X) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < N) {
    X[x] = fmaxf(fminf(X[x], max_value), min_value);
  }
}

template <>
__global__ void clip_kernel<double>(
  const int N, const double min_value, const double max_value, double *X) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < N) {
    X[x] = fmax(fmin(X[x], max_value), min_value);
  }
}
template<typename Dtype>
void caffe_gpu_clip(const int N, Dtype min_value, Dtype max_value, Dtype *X) {
  clip_kernel<Dtype> <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
    N, min_value, max_value, X);
}

KERNEL_FUNC(
  asum_kernel_0,
  asum[i] += fabs(in[i * N + j]),
  const Dtype *in, Dtype *asum);
KERNEL_FUNC(
  asum_kernel_1,
  asum[j] += fabs(in[i * N + j]),
  const Dtype *in, Dtype *asum);

KERNEL_FUNC(
  set_kernel_0,
  out[i * N + j] = in[i * N + j] >= 0 ? asum[j] : -asum[j],
  const Dtype *in, const Dtype *asum, Dtype *out);
KERNEL_FUNC(
  set_kernel_1,
  out[i * N + j] = in[i * N + j] >= 0 ? asum[i] : -asum[i],
  const Dtype *in, const Dtype *asum, Dtype *out);

template <typename Dtype>
void caffe_gpu_binary_approx(
  const int axis, const int M, const int N, const Dtype* in,
  Dtype* out, Dtype *scale) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    caffe_gpu_set<Dtype>(M, Dtype(0), scale);
    asum_kernel_0<Dtype> <<< blocks, threads>>> (M, N, in, scale);
    caffe_gpu_scal<Dtype>(M, Dtype(1. / N), scale);
    set_kernel_0<Dtype> <<< blocks, threads>>> (M, N, in, scale, out);
  }
  else {
    caffe_gpu_set<Dtype>(N, Dtype(0), scale);
    asum_kernel_1<Dtype> <<< blocks, threads>>> (M, N, in, scale);
    caffe_gpu_scal<Dtype>(N, Dtype(1. / M), scale);
    set_kernel_1<Dtype> <<< blocks, threads>>>(M, N, in, scale, out);
  }
}

KERNEL_FUNC(
ternary_kernel_0,  {
  if (in[i * N + j] > delta[i]) {
    out[i * N + j] = Dtype(1);
    scale[i] += in[i * N + j];
    ++sum[i];
  }
  else if (in[i * N + j] < -delta[i]) {
    out[i * N + j] = Dtype(-1);
    scale[i] -= in[i * N + j];
    ++sum[i];
  }
  else
    out[i * N + j] = 0;
},
const Dtype *in, const Dtype *delta, Dtype *scale,
Dtype *sum, Dtype *out);

KERNEL_FUNC(
ternary_kernel_1, {
  if (in[i * N + j] > delta[j]) {
    out[i * N + j] = Dtype(1);
    scale[j] += in[i * N + j];
    ++sum[j];
  }
  else if (in[i * N + j] < -delta[i]) {
    out[i * N + j] = Dtype(-1);
    scale[j] -= in[i * N + j];
    ++sum[j];
  }
  else
    out[i * N + j] = 0;
},
const Dtype *in, const Dtype *delta, Dtype *scale,
Dtype* sum, Dtype *out);

KERNEL_FUNC(
  mul_eq_kernel_0,
  out[i * N + j] *= scale[i],
  const Dtype *scale, Dtype *out);

KERNEL_FUNC(
  mul_eq_kernel_1,
  out[i * N + j] *= scale[j],
  const Dtype *scale, Dtype *out);

template <typename Dtype>
__global__ void div_eq_kernel(const int N, const Dtype *X, Dtype* Y) {
  int i = threadIdx.x;
  if (i < N) {
    if (X[i] > 0)
      Y[i] /= X[i];
  }
}
template <typename Dtype>
void caffe_gpu_ternary_approx(
  const int axis, const int M, const int N, const Dtype *in,
  Dtype* out, Dtype *scale, Dtype *delta, Dtype *sum) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    caffe_gpu_set<Dtype>(M, Dtype(0), scale);
    caffe_gpu_set<Dtype>(M, Dtype(0), delta);
    caffe_gpu_set<Dtype>(M, Dtype(0), sum);
    asum_kernel_0<Dtype> <<<blocks, threads>>> (M, N, in, delta);
    caffe_gpu_scal<Dtype>(M, Dtype(0.7 / N), delta);
    ternary_kernel_0<Dtype> <<<blocks, threads>>>(
      M, N, in, delta, scale, sum, out);
    div_eq_kernel<Dtype> <<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>(
      M, sum, scale);
    mul_eq_kernel_0<Dtype> <<<blocks, threads>>> (M, N, scale, out);
  }
  else {
    caffe_gpu_set<Dtype>(N, Dtype(0), scale);
    caffe_gpu_set<Dtype>(N, Dtype(0), delta);
    caffe_gpu_set<Dtype>(N, Dtype(0), sum);
    asum_kernel_1<Dtype> <<<blocks, threads>>> (M, N, in, delta);
    caffe_gpu_scal<Dtype>(M, Dtype(0.7 / M), delta);
    ternary_kernel_1<Dtype> <<<blocks, threads>>>(
      M, N, in, delta, scale, sum, out);
    div_eq_kernel<Dtype> <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, sum, scale);
    mul_eq_kernel_0<Dtype> <<<blocks, threads>>> (M, N, scale, out);
  }
}

#define INSTANTIATE_BINARY_MATH(Dtype) \
  template void caffe_gpu_binary_gradient<Dtype>( \
      const int axis, const int M, const int N, \
      const Dtype *in, const Dtype *scale, Dtype *grad);  \
  \
  template void caffe_gpu_ternary_gradient<Dtype>(  \
      const int axis, const int M, const int N, \
      const Dtype *in, const Dtype *scale, const Dtype *delta, Dtype *grad);  \
  \
  template void caffe_gpu_clip<Dtype>(  \
      const int N, Dtype min_value, Dtype max_value, Dtype *X); \
  \
  template void caffe_gpu_binary_approx<Dtype>( \
      const int axis, const int M, const int N, const Dtype* in,  \
      Dtype* out, Dtype *scale);  \
  \
  template void caffe_gpu_ternary_approx<Dtype>(  \
      const int axis, const int M, const int N, const Dtype *in,  \
      Dtype* out, Dtype *scale, Dtype *delta, Dtype *sum);

INSTANTIATE_BINARY_MATH(float);
INSTANTIATE_BINARY_MATH(double);
#endif // CPU_ONLY
}
