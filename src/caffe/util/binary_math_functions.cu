#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
#ifndef CPU_ONLY

#define KERNEL_FUNC(NAME, OP, PARAMS...) \
  template <typename Dtype> \
  __global__ void NAME(const int M, const int N, PARAMS)\
  { \
    int i = blockIdx.x * blockDim.x + threadIdx.x; \
    int j = blockIdx.y * blockDim.y + threadIdx.y; \
    if (i < M && j < N) { \
      OP; \
    } \
  }

KERNEL_FUNC(
  binary_gradient_kernel_0,
  grad[i * N + j] *=
    mul + (fabs(in[i * N + j]) <= Dtype(1) ? scale[i] : Dtype(0)),
  const Dtype *in, const Dtype *scale, Dtype *grad, const Dtype mul);
KERNEL_FUNC(
  binary_gradient_kernel_1,
  grad[i * N + j] *=
    mul + (fabs(in[i * N + j]) <= Dtype(1) ? scale[j] : Dtype(0)),
  const Dtype *in, const Dtype *scale, Dtype *grad, const Dtype mul);

template<typename Dtype>
void caffe_gpu_binary_gradient(
  const int axis, const int M, const int N,
  const Dtype *in, const Dtype *scale, Dtype *grad) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    const Dtype mul = 1. / N;
    binary_gradient_kernel_0<Dtype> <<< blocks, threads>>> (
      M, N, in, scale, grad, mul);
  }
  else {
    const Dtype mul = 1. / M;
    binary_gradient_kernel_1<Dtype> <<< blocks, threads>>> (
      M, N, in, scale, grad, mul);
  }
}

template<typename Dtype>
__global__ void ternary_gradient_kernel_0(
  const int M, const int N, const Dtype *in, const Dtype *scale,
  const Dtype *delta, Dtype *grad, const Dtype mul) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < M && y < N) {
    register Dtype val = fabs(in[x * N + y]);
    grad[x * N + y] *= mul + Dtype(val <= delta[x] ? 1 : val > Dtype(1) ? 0 :
                                   scale[x]);
  }
}

template<typename Dtype>
__global__ void ternary_gradient_kernel_1(
  const int M, const int N, const Dtype *in, const Dtype *scale,
  const Dtype *delta, Dtype *grad, const Dtype mul) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < M && y < N) {
    register Dtype val = fabs(in[x * N + y]);
    grad[x * N + y] *= mul + Dtype(val <= delta[y] ? 1 : val > Dtype(1) ? 0 :
                                   scale[y]);
  }
}

template<typename Dtype>
void caffe_gpu_ternary_gradient(
  const int axis, const int M, const int N,
  const Dtype *in, const Dtype *scale, const Dtype *delta, Dtype *grad) {
  dim3 blocks((M - 1) / 32 + 1, (N - 1) / 16 + 1, 1);
  dim3 threads(32, 16, 1);
  if (axis == 0) {
    const Dtype mul = 1. / N;
    ternary_gradient_kernel_0<Dtype> <<< blocks, threads>>> (
      M, N, in, scale, delta, grad, mul);
  }
  else {
    const Dtype mul = 1. / M;
    ternary_gradient_kernel_1<Dtype> <<< blocks, threads>>> (
      M, N, in, scale, delta, grad, mul);
  }
}

template <typename Dtype>
__global__ void clip_kernel(
  const int N, const Dtype min_value, const Dtype max_value, Dtype *X);
template <>
__global__ void clip_kernel<float>(
  const int N, const float min_value, const float max_value, float *X) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < N) {
    X[x] = fmaxf(fminf(X[x], max_value), min_value);
  }
}

template <>
__global__ void clip_kernel<double>(
  const int N, const double min_value, const double max_value, double *X) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < N) {
    X[x] = fmax(fmin(X[x], max_value), min_value);
  }
}
template<typename Dtype>
void caffe_gpu_clip(const int N, Dtype min_value, Dtype max_value, Dtype *X) {
  clip_kernel<Dtype> <<< CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
    N, min_value, max_value, X);
}

template <typename Dtype>
__global__ void binary_approx_kernel_0(
  const int M, const int N,  bool use_bias,
  const Dtype* in, Dtype* out, Dtype *scale) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= M) return ;
  scale[i] = 0;
  for (int j = 0; j < N; ++j)
    scale[i] += fabs(in[i * N + j]);
  scale[i] /= Dtype(N);
  for (int j = 0; j < N; ++j)
    out[i * N + j] = in[i * N + j] >= 0 ? scale[i] : -scale[i];
}

template <typename Dtype>
__global__ void binary_approx_kernel_1(
  const int M, const int N,  bool use_bias,
  const Dtype* in, Dtype* out, Dtype *scale) {
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j >= N) return ;
  scale[j] = 0;
  for (int i = 0; i < M; ++i)
    scale[j] += fabs(in[i * N + j]);
  scale[j] /= Dtype(M);
  for (int i = 0; i < M; ++i)
    out[i * N + j] = in[i * N + j] >= 0 ? scale[j] : -scale[j];
}

template <typename Dtype>
void caffe_gpu_binary_approx(
  const int axis, const int M, const int N,  bool use_bias,
  const Dtype* in, Dtype* out, Dtype *scale) {
  if (axis == 0) {
    binary_approx_kernel_0<Dtype>
    <<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>
    (M, N, use_bias, in, out, scale);
  }
  else {
    binary_approx_kernel_1<Dtype>
    <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>
    (M, N, use_bias, in, out, scale);
  }
}

template <typename Dtype>
__global__ void ternary_approx_kernel_0(
  const int M, const int N, bool use_bias,
  const Dtype * in, Dtype * out, Dtype * scale, Dtype * delta, Dtype * sum) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= M) return ;
  scale[i] = 0;
  delta[i] = 0;
  sum[i] = 0;
  for (int j = 0; j < N; ++j)
    delta[i] += fabs(in[i * N + j]);
  delta[i] *= 0.7 / N;
  for (int j = 0; j < N; ++j) {
    Dtype val = fabs(in[i * N + j]);
    if (val > delta[i]) {
      scale[i] += val;
      ++sum[i];
    }
  }
  if (sum[i] > 0) scale[i] /= sum[i];
  for (int j = 0; j < N; ++j) {
    if (in[i * N + j] > delta[i])
      out[i * N + j] = scale[i];
    else if (in[i * N + j] < -delta[i])
      out[i * N + j] = -scale[i];
    else
      out[i * N + j] = 0;
  }
}

template <typename Dtype>
__global__ void ternary_approx_kernel_1(
  const int M, const int N, bool use_bias,
  const Dtype * in, Dtype * out, Dtype * scale, Dtype * delta, Dtype * sum) {
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j >= N) return ;
  delta[j] = 0;
  scale[j] = 0;
  sum[j] = 0;
  for (int i = 0; i < M; ++i)
    delta[j] += fabs(in[i * N + j]);
  delta[j] *= 0.7 / M;
  for (int i = 0; i < M; ++i) {
    Dtype val = fabs(in[i * N + j]);
    if (val > delta[j]) {
      scale[j] += val;
      ++sum[j];
    }
  }
  if (sum[j] > 0) scale[j] /= sum[j];
  for (int i = 0; i < M; ++i) {
    if (in[i * N + j] > delta[j])
      out[i * N + j] = scale[j];
    else if (in[i * N + j] < -delta[j])
      out[i * N + j] = -scale[j];
    else
      out[i * N + j] = 0;
  }
}
template <typename Dtype>
void caffe_gpu_ternary_approx(
  const int axis, const int M, const int N, bool use_bias,
  const Dtype * in, Dtype * out, Dtype * scale, Dtype * delta, Dtype * sum) {
  if (axis == 0) {
    ternary_approx_kernel_0<Dtype>
    <<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>
    (M, N, use_bias, in, out, scale, delta, sum);
  }
  else {
    ternary_approx_kernel_1<Dtype>
    <<<CAFFE_GET_BLOCKS(M), CAFFE_CUDA_NUM_THREADS>>>
    (M, N, use_bias, in, out, scale, delta, sum);
  }
}
#define INSTANTIATE_BINARY_MATH(Dtype) \
  template void caffe_gpu_binary_gradient<Dtype>( \
      const int axis, const int M, const int N, \
      const Dtype *in, const Dtype *scale, Dtype *grad);  \
  \
  template void caffe_gpu_ternary_gradient<Dtype>(  \
      const int axis, const int M, const int N, \
      const Dtype *in, const Dtype *scale, const Dtype *delta, Dtype *grad);  \
  \
  template void caffe_gpu_clip<Dtype>(  \
      const int N, Dtype min_value, Dtype max_value, Dtype *X); \
  \
  template void caffe_gpu_binary_approx<Dtype>( \
      const int axis, const int M, const int N, bool use_bias,  \
      const Dtype* in, Dtype* out, Dtype *scale);  \
  \
  template void caffe_gpu_ternary_approx<Dtype>(  \
      const int axis, const int M, const int N, bool use_bias, \
      const Dtype *in, Dtype* out, Dtype *scale, Dtype *delta, Dtype *sum);

INSTANTIATE_BINARY_MATH(float);
INSTANTIATE_BINARY_MATH(double);
#endif // CPU_ONLY
}
