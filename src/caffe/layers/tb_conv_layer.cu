#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/tb_conv_layer.hpp"
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
void __global__ scale_kernel_1(
    const int n, const int width, const Dtype* alpha, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = out[index] * alpha[index / width]; }
}
template <typename Dtype>
void __global__ scale_kernel_2(
    const int n, const int width, const Dtype* beta, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = out[index] * beta[index % width]; }
}

template <typename Dtype>
__global__ void conv2D_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, const int height_col,
    const int width_col, Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index   = index / width_col;
    const int h_col     = h_index % height_col;
    const int w_col     = index % width_col;
    const int c_im      = h_index / height_col;
    const int h_offset  = h_col * stride_h - pad_h;
    const int w_offset  = w_col * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (c_im * height_col + h_col) * width_col + w_col;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    *data_col_ptr = 0;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr +=
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                ? data_im_ptr[i * dilation_h * width + j * dilation_w]
                : 0;
      }
    }
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::conv2D_gpu(
    const Dtype* in, const int num, Dtype* out) {
  const int height     = conv_input_shape_.cpu_data()[1];
  const int width      = conv_input_shape_.cpu_data()[2];
  const int kernel_h   = kernel_shape_.cpu_data()[0];
  const int kernel_w   = kernel_shape_.cpu_data()[1];
  const int pad_h      = pad_.cpu_data()[0];
  const int pad_w      = pad_.cpu_data()[1];
  const int stride_h   = stride_.cpu_data()[0];
  const int stride_w   = stride_.cpu_data()[1];
  const int dilation_h = dilation_.cpu_data()[0];
  const int dilation_w = dilation_.cpu_data()[1];

  int height_col =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = num * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  conv2D_kernel<Dtype>
      <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(num_kernels,
          in, height, width, kernel_h, kernel_w, pad_h, pad_w, stride_h,
          stride_w, dilation_h, dilation_w, height_col, width_col, out);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::forward_gpu_gemm(
    const Dtype* input, const Dtype* beta, const Dtype* weight, Dtype* output) {
  const Dtype* col_buff = input;
  if (!is_1x1_) {
    conv_im2col_gpu(input, col_buffer_.mutable_gpu_data());
    Dtype* temp = col_buffer_.mutable_gpu_data();
    for (int g = 0; g < group_; ++g) {
      scale_kernel_2<Dtype>
          <<<CAFFE_GET_BLOCKS(col_offset_), CAFFE_CUDA_NUM_THREADS>>>(
              col_offset_, out_spatial_dim_, beta + out_spatial_dim_ * g,
              temp + col_offset_ * g);
    }
    col_buff = col_buffer_.gpu_data();
  }
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, out_channels_,
        out_spatial_dim_, kernel_dim_, (Dtype) 1., weight + weight_offset_ * g,
        col_buff + col_offset_ * g, (Dtype) 0., output + output_offset_ * g);
  }
}
template <typename Dtype>
void TBConvolutionLayer<Dtype>::backward_gpu_gemm(
    const Dtype* output, const Dtype* weights, Dtype* input) {
  Dtype* col_buff = col_buffer_.mutable_gpu_data();
  if (is_1x1_) {
    col_buff = input;
  }
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, kernel_dim_,
        out_spatial_dim_, out_channels_, (Dtype) 1.,
        weights + weight_offset_ * g, output + output_offset_ * g, (Dtype) 0.,
        col_buff + col_offset_ * g);
  }
  if (!is_1x1_) {
    conv_col2im_gpu(col_buff, input);
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::weight_gpu_gemm(const Dtype* input,
    const Dtype* output, const Dtype* beta, Dtype* weight_diff) {
  const Dtype* col_buff = input;
  if (!is_1x1_) {
    conv_im2col_gpu(input, col_buffer_.mutable_gpu_data());
    Dtype* temp = col_buffer_.mutable_gpu_data();
    for (int g = 0; g < group_; ++g) {
      scale_kernel_2<Dtype>
          <<<CAFFE_GET_BLOCKS(col_offset_), CAFFE_CUDA_NUM_THREADS>>>(
              col_offset_, out_spatial_dim_, beta + out_spatial_dim_ * g,
              temp + col_offset_ * g);
    }
    col_buff = col_buffer_.gpu_data();
  }
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, out_channels_, kernel_dim_,
        out_spatial_dim_, (Dtype) 1., output + output_offset_ * g,
        col_buff + col_offset_ * g, (Dtype) 1.,
        weight_diff + weight_offset_ * g);
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::backward_gpu_bias(
    Dtype* bias, const Dtype* input) {
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_output_, out_spatial_dim_, 1., input,
      bias_multiplier_.gpu_data(), 1., bias);
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* alpha   = this->blobs_[1]->gpu_data();
  const Dtype* b1_data = bottom[1]->gpu_data();
  const Dtype* b2_data = bottom[2]->gpu_data();
  int count            = this->blobs_[0]->count();

  caffe_gpu_clip<Dtype>(count, -1, 1, this->blobs_[0]->mutable_gpu_data());
  caffe_gpu_sign<Dtype>(
      count, this->blobs_[0]->gpu_data(), weight_.mutable_gpu_data());
  scale_kernel_1<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, kernel_dim_, alpha, weight_.mutable_gpu_data());

  if (!is_1x1_) {
    conv2D_gpu(b1_data, num_ * group_, beta_.mutable_gpu_data());
    conv2D_gpu(b2_data, num_ * group_, sum_.mutable_gpu_data());
  } else {
    caffe_copy<Dtype>(beta_.count(), b1_data, beta_.mutable_gpu_data());
    caffe_copy<Dtype>(sum_.count(), b2_data, sum_.mutable_gpu_data());
  }
  caffe_gpu_div<Dtype>(out_spatial_dim_, beta_.gpu_data(), sum_.gpu_data(),
      beta_.mutable_gpu_data());

  const Dtype* beta = beta_.gpu_data();
  if (is_1x1_) {
    Dtype* bottom_data = bottom[0]->mutable_gpu_data();
    for (int i = 0; i < num_ * group_; ++i) {
      scale_kernel_2<Dtype>
          <<<CAFFE_GET_BLOCKS(col_offset_), CAFFE_CUDA_NUM_THREADS>>>(
              col_offset_, out_spatial_dim_, beta + out_spatial_dim_ * i,
              bottom_data + col_offset_ * i);
    }
  }

  const Dtype* weight      = weight_.gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data          = top[0]->mutable_gpu_data();
  for (int n = 0; n < num_; ++n) {
    forward_gpu_gemm(bottom_data + n * bottom_dim_, beta + n * beta_dim_,
        weight, top_data + n * top_dim_);
  }

  if (this->bias_term_) {
    const Dtype* bias = this->blobs_[2]->gpu_data();
    top_data          = top[0]->mutable_gpu_data();
    for (int n = 0; n < num_; ++n) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_output_,
          out_spatial_dim_, 1, (Dtype) 1., bias, bias_multiplier_.gpu_data(),
          (Dtype) 1., top_data + n * top_dim_);
    }
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight      = weight_.gpu_data();
  const Dtype* alpha       = this->blobs_[1]->gpu_data();
  const Dtype* top_diff    = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* beta        = is_1x1_ ? bottom[1]->gpu_data() : beta_.gpu_data();
  Dtype* weight_diff       = this->blobs_[0]->mutable_gpu_diff();
  Dtype* bottom_diff       = bottom[0]->mutable_gpu_diff();

  // Bias gradient, if necessary.
  if (bias_term_ && this->param_propagate_down_[2]) {
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    for (int n = 0; n < this->num_; ++n) {
      this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
    }
  }
  // gradient w.r.t. weight. Note that we will accumulate diffs.
  if (this->param_propagate_down_[0]) {
    for (int n = 0; n < num_; ++n) {
      this->weight_gpu_gemm(bottom_data + n * bottom_dim_,
          top_diff + n * top_dim_, beta + n * beta_dim_, weight_diff);
    }
  }
  // gradient w.r.t. bottom data, if necessary.
  if (propagate_down[0]) {
    for (int n = 0; n < num_; ++n) {
      this->backward_gpu_gemm(
          top_diff + n * top_dim_, weight, bottom_diff + n * bottom_dim_);
    }
  }
  weight_diff       = this->blobs_[0]->mutable_gpu_diff();
  Dtype* alpha_diff = this->blobs_[1]->mutable_gpu_diff();
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num_output_, kernel_dim_, (Dtype) 1.,
      weight_diff, sum_multiplier_.gpu_data(), (Dtype) 0., alpha_diff);
  const int count = num_output_ * kernel_dim_;
  scale_kernel_1<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, kernel_dim_, alpha, weight_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(TBConvolutionLayer);
}  // namespace caffe
