#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/tb_conv_layer.hpp"
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
void __global__ scale_kernel_1(
    const int n, const int width, const Dtype* alpha, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = out[index] * alpha[index / width]; }
}
template <typename Dtype>
void __global__ scale_kernel_2(
    const int n, const int width, const Dtype* beta, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = out[index] * beta[index % width]; }
}
template <typename Dtype>
void __global__ backward_scale_kernel(
    const int n, const int width, const Dtype* alpha, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = out[index] / alpha[index / width]; }
}
template <typename Dtype>
__global__ void conv2D_kernel(
    const int n, const Dtype* data_im, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index   = index / width_col;
    const int h_col     = h_index % height_col;
    const int w_col     = index % width_col;
    const int c_im      = h_index / height_col;
    const int h_offset  = h_col * stride_h - pad_h;
    const int w_offset  = w_col * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (c_im * height_col + h_col) * width_col + w_col;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    *data_col_ptr = 0;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr +=
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                ? data_im_ptr[i * dilation_h * width + j * dilation_w]
                : 0;
      }
    }
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::conv2D_gpu(
    const Dtype* in, const int num, Dtype* out) {
  const int height     = conv_input_shape_.cpu_data()[1];
  const int width      = conv_input_shape_.cpu_data()[2];
  const int kernel_h   = kernel_shape_.cpu_data()[0];
  const int kernel_w   = kernel_shape_.cpu_data()[1];
  const int pad_h      = pad_.cpu_data()[0];
  const int pad_w      = pad_.cpu_data()[1];
  const int stride_h   = stride_.cpu_data()[0];
  const int stride_w   = stride_.cpu_data()[1];
  const int dilation_h = dilation_.cpu_data()[0];
  const int dilation_w = dilation_.cpu_data()[1];

  int height_col =
      (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = num * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  conv2D_kernel<Dtype>
      <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
          num_kernels, in, height, width, kernel_h, kernel_w, pad_h, pad_w,
          stride_h, stride_w, dilation_h, dilation_w, height_col, width_col,
          out);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::backward_gpu_gemm(
    const Dtype* output, const Dtype* weights, Dtype* input) {
  Dtype* col_buff = col_buffer_.mutable_gpu_data();
  if (is_1x1_) {
    col_buff = input;
  }
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(
        CblasTrans, CblasNoTrans, kernel_dim_, out_spatial_dim_, out_channels_,
        (Dtype) 1., weights + weight_offset_ * g, output + output_offset_ * g,
        (Dtype) 0., col_buff + col_offset_ * g);
  }
  if (!is_1x1_) {
    conv_col2im_gpu(col_buff, input);
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::weight_gpu_gemm(
    const Dtype* input, const Dtype* output, const Dtype* beta,
    Dtype* weight_diff) {
  const Dtype* col_buff = input;
  if (!is_1x1_) {
    conv_im2col_gpu(input, col_buffer_.mutable_gpu_data());
    col_buff = col_buffer_.gpu_data();

    for (int g = 0; g < group_; ++g) {
      scale_kernel_2<Dtype>
          <<<CAFFE_GET_BLOCKS(col_offset_), CAFFE_CUDA_NUM_THREADS>>>(
              col_offset_, out_spatial_dim_, beta + out_spatial_dim_ * g,
              col_buffer_.mutable_gpu_data() + col_offset_ * g);
    }
  }
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(
        CblasNoTrans, CblasTrans, out_channels_, kernel_dim_, out_spatial_dim_,
        (Dtype) 1., output + output_offset_ * g, col_buff + col_offset_ * g,
        (Dtype) 1., weight_diff + weight_offset_ * g);
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::backward_gpu_bias(
    Dtype* bias, const Dtype* input) {
  caffe_gpu_gemv<Dtype>(
      CblasNoTrans, num_output_, out_spatial_dim_, 1., input,
      bias_multiplier_.gpu_data(), 1., bias);
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bias  = this->blobs_[2]->gpu_data();
  const Dtype* alpha = this->blobs_[1]->gpu_data();
  Dtype* col_buff    = top[0]->mutable_gpu_data();
  Dtype* weight_data = this->blobs_[0]->mutable_gpu_data();
  Dtype* weight      = weight_.mutable_gpu_data();
  Dtype* beta        = bottom[1]->mutable_gpu_data();
  Dtype* sum         = bottom[2]->mutable_gpu_data();
  int count          = this->blobs_[0]->count();

  caffe_gpu_clip<Dtype>(count, -1, 1, weight_data);
  caffe_gpu_sign<Dtype>(count, weight_data, weight_.mutable_gpu_data());
  scale_kernel_1<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, kernel_dim_, alpha, weight);
  if (!is_1x1_) {
    conv2D_gpu(beta, num_ * group_, beta_.mutable_gpu_data());
    conv2D_gpu(sum, num_ * group_, sum_.mutable_gpu_data());
    col_buff = col_buffer_.mutable_gpu_data();
    beta     = beta_.mutable_gpu_data();
    sum      = sum_.mutable_gpu_data();
  }
  caffe_gpu_div<Dtype>(bottom[1]->count(), beta, sum, beta);
  for (int n = 0; n < num_; ++n) {
    const Dtype* bottom_data = bottom[0]->gpu_data() + n * bottom_dim_;
    Dtype* top_data          = top[0]->mutable_gpu_data() + n * this->top_dim_;

    if (!is_1x1_) {
      conv_im2col_gpu(bottom_data, col_buff);
    }
    for (int g = 0; g < group_; ++g) {
      scale_kernel_2<Dtype>
          <<<CAFFE_GET_BLOCKS(col_offset_), CAFFE_CUDA_NUM_THREADS>>>(
              col_offset_, out_spatial_dim_,
              beta + (n * group_ + g) * out_spatial_dim_,
              col_buff + col_offset_ * g);

      caffe_gpu_gemm<Dtype>(
          CblasNoTrans, CblasNoTrans, out_channels_, out_spatial_dim_,
          kernel_dim_, (Dtype) 1., weight + weight_offset_ * g,
          col_buff + col_offset_ * g, (Dtype) 0.,
          top_data + output_offset_ * g);
    }
    if (this->bias_term_) {
      caffe_gpu_gemm<Dtype>(
          CblasNoTrans, CblasNoTrans, num_output_, out_spatial_dim_, 1,
          (Dtype) 1., bias, bias_multiplier_.gpu_data(), (Dtype) 1., top_data);
    }
  }
}

template <typename Dtype>
void TBConvolutionLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = weight_.gpu_data();
  const Dtype* alpha  = this->blobs_[1]->gpu_data();
  Dtype* weight_diff  = this->blobs_[0]->mutable_gpu_diff();
  Dtype* alpha_diff   = this->blobs_[1]->mutable_gpu_diff();

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (bias_term_ && this->param_propagate_down_[2]) {
      Dtype* bias_diff = this->blobs_[2]->mutable_gpu_diff();
      for (int n = 0; n < num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff       = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(
              bottom_data + n * bottom_dim_, top_diff + n * top_dim_,
              beta_.gpu_data() + n * group_ * out_spatial_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(
              top_diff + n * top_dim_, weight, bottom_diff + n * bottom_dim_);
        }
      }
    }
  }
  caffe_gpu_gemv<Dtype>(
      CblasNoTrans, num_output_, kernel_dim_, (Dtype) 1., weight_diff,
      sum_multiplier_.gpu_data(), (Dtype) 0., alpha_diff);
  const int count = num_output_ * kernel_dim_;
  backward_scale_kernel<Dtype>
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, kernel_dim_, alpha, weight_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(TBConvolutionLayer);
}  // namespace caffe
