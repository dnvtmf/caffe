#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit

#include <thrust/functional.h>
#include "caffe/layers/ternary_layer.hpp"
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/cuda_reduce.hpp"

namespace caffe {

inline __device__ float gpu_abs(float x) { return fabsf(x); }
inline __device__ double gpu_abs(double x) { return fabs(x); }

template <typename Dtype>
void __global__ beta_div_add_kernel(
    const int n, const Dtype *sum, const Dtype add_value, Dtype *beta) {
  CUDA_KERNEL_LOOP(index, n) {
    if (sum[index] > 0) beta[index] /= sum[index];
    beta[index] += add_value;
  }
}

/**
\delta_c = \frac{t}{num * dim} \sum_{n=1}^{num}{\sum_{i=1}^{dim}{|in[n][c][i]|}}
*/
template <typename Dtype>
void __global__ delta_kernel(const int num, const int channels, const int dim,
    const Dtype threshold_t, const Dtype *in, Dtype *delta) {
  const int c  = blockIdx.x;
  const int id = threadIdx.x;
  volatile __shared__ Dtype temp[CAFFE_CUDA_NUM_THREADS - WARP_SIZE];
  Dtype val = 0;
  in += c * dim;
  for (int n = 0; n < num; ++n) {
    for (int j = id; j < dim; j += blockDim.x) {
      val += gpu_abs(in[j]);
    }
    in += channels * dim;
  }
  if (id >= WARP_SIZE) temp[id - WARP_SIZE] = val;
  __syncthreads();
  if (id < WARP_SIZE) {
#pragma unroll
    for (int k = id; k < (CAFFE_CUDA_NUM_THREADS - WARP_SIZE); k += WARP_SIZE)
      val += temp[k];
    temp[id] = val;
  }
  // __syncthreads();
  if (id == 0) {
    for (int k = 1; k < WARP_SIZE; ++k) val += temp[k];
    delta[c]   = val * threshold_t;
  }
}

template <typename Dtype>
void __global__ forward_kernel(const int n, const int channels, const int dim,
    const Dtype *delta, const Dtype *in, Dtype *out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int c = index / dim % channels;
    out[index]  = in[index] > delta[c] ? 1 : (in[index] < -delta[c] ? -1 : 0);
  }
}

template <typename Dtype>
void __global__ backward_kernel(const int n, const int channels,
    const int group_channels, const int dim, const Dtype *delta,
    const Dtype *beta, const Dtype *in, Dtype *out) {
  CUDA_KERNEL_LOOP(index, n) {
    int y      = index % dim;
    int index2 = index / dim;
    int c      = index2 % channels;
    index2     = (index2 / group_channels) * dim + y;
    if (gpu_abs(in[index]) > delta[c]) out[index] *= beta[index2];
  }
}

template <typename Dtype>
void TernaryLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  const int count = bottom[0]->count();
  if (!use_global_stats_) {
    Dtype threshold_t = threshold_t_ / Dtype(count / channels_);
    delta_kernel<Dtype><<<channels_, CAFFE_CUDA_NUM_THREADS>>>(num_, channels_,
        dim_, threshold_t, bottom[0]->gpu_data(), delta_.mutable_gpu_data());
    caffe_gpu_axpby<Dtype>(channels_, 1. - moving_average_fraction_,
        delta_.gpu_data(), moving_average_fraction_,
        this->blobs_[0]->mutable_gpu_data());
  }

  const Dtype *delta =
      use_global_stats_ ? this->blobs_[0]->gpu_data() : delta_.gpu_data();
  forward_kernel<Dtype>
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, channels_,
          dim_, delta, bottom[0]->gpu_data(), top[0]->mutable_gpu_data());
  if (scale_term_) {
    caffe_gpu_input_scale<Dtype>(num_ * group_, channels_ / group_, dim_,
        bottom[0]->gpu_data(), top[0]->gpu_data(), top[1]->mutable_gpu_data(),
        top[2]->mutable_gpu_data());
  }
}

template <typename Dtype>
void TernaryLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
    const vector<bool> &propagate_down, const vector<Blob<Dtype> *> &bottom) {
  if (propagate_down[0]) {
    const int count = bottom[0]->count();
    caffe_copy(count, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff());
    if (scale_term_) {
      beta_div_add_kernel<Dtype>
          <<<CAFFE_GET_BLOCKS(top[1]->count()), CAFFE_CUDA_NUM_THREADS>>>(
              top[1]->count(), top[2]->gpu_data(),
              Dtype(1.) / Dtype(channels_ / group_),
              top[1]->mutable_gpu_data());
      backward_kernel<Dtype>
          <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
              channels_, channels_ / group_, dim_, delta_.gpu_data(),
              top[1]->gpu_data(), bottom[0]->gpu_data(),
              bottom[0]->mutable_gpu_diff());
    }
    caffe_gpu_clip_grad(
        count, bottom[0]->gpu_data(), bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TernaryLayer);
}  // namespace caffe
