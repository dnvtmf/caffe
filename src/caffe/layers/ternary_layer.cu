#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit

#include <thrust/functional.h>
#include "caffe/layers/ternary_layer.hpp"
#include "caffe/util/binary_math_functions.hpp"
#include "caffe/util/cuda_reduce.hpp"

namespace caffe {

inline __device__ float gpu_abs(float x) { return fabsf(x); }
inline __device__ double gpu_abs(double x) { return fabs(x); }

/**
\delta_c = \frac{t}{num * dim} \sum_{n=1}^{num}{\sum_{i=1}^{dim}{|in[n][c][i]|}}
*/
template <typename Dtype>
void __global__ delta_kernel(const int num, const int channels, const int dim,
    const Dtype threshold_t, const Dtype *in, Dtype *delta) {
  const int c  = blockIdx.x;
  const int id = threadIdx.x;
  volatile __shared__ Dtype temp[CAFFE_CUDA_NUM_THREADS - WARP_SIZE];
  Dtype val = 0;
  in += c * dim;
  for (int n = 0; n < num; ++n) {
    for (int j = id; j < dim; j += blockDim.x) {
      val += gpu_abs(in[j]);
    }
    in += channels * dim;
  }
  if (id >= WARP_SIZE) temp[id - WARP_SIZE] = val;
  __syncthreads();
  if (id < WARP_SIZE) {
#pragma unroll
    for (int k = id; k < (CAFFE_CUDA_NUM_THREADS - WARP_SIZE); k += WARP_SIZE)
      val += temp[k];
    temp[id] = val;
  }
  __syncthreads();
  if (id == 0) {
    for (int k = 1; k < WARP_SIZE; ++k) val += temp[k];
    delta[c]   = val * threshold_t;
  }
}

template <typename Dtype>
void __global__ forward_kernel(const int channels, const int dim,
    const Dtype *delta, const Dtype *in, Dtype *out, Dtype *beta, Dtype *sum) {
  const int idx = blockIdx.x;
  const int id  = threadIdx.x;
  volatile __shared__ Dtype temp[CAFFE_CUDA_NUM_THREADS - WARP_SIZE];
  volatile __shared__ Dtype temp2[CAFFE_CUDA_NUM_THREADS - WARP_SIZE];
  Dtype val = 0, val2 = 0;
  for (int c = id; c < channels; c += blockDim.x) {
    const int offset = c * dim + idx;
    out[offset]      = 0;
    if (in[offset] > delta[c]) {
      out[offset] = 1;
      val += in[offset];
      val2++;
    }
    if (in[offset] < -delta[c]) {
      out[offset] = -1;
      val -= in[offset];
      val2++;
    }
  }
  if (id >= WARP_SIZE) {
    temp[id - WARP_SIZE]  = val;
    temp2[id - WARP_SIZE] = val2;
  }
  __syncthreads();
  if (id < WARP_SIZE) {
#pragma unroll
    for (int k = id; k < (CAFFE_CUDA_NUM_THREADS - WARP_SIZE); k += WARP_SIZE)
      val += temp[k], val2 += temp2[k];
    temp[id]  = val;
    temp2[id] = val2;
  }
  __syncthreads();
  if (id == 0) {
    for (int k = 1; k < WARP_SIZE; ++k) val += temp[k], val2 += temp2[k];
    beta[idx]  = val;
    sum[idx]   = val2;
  }
}

template <typename Dtype>
void __global__ backward_kernel(const int n, const int channels,
    const int group_channels, const int dim, const Dtype *delta,
    const Dtype *in, const Dtype *beta, Dtype *out) {
  CUDA_KERNEL_LOOP(index, n) {
    int y      = index % dim;
    int index2 = index / dim;
    int c      = index2 % channels;
    index2     = (index2 / group_channels) * dim + y;
    if (gpu_abs(in[index]) > delta[c]) out[index] *= beta[index2];
    if (gpu_abs(in[index]) > 1) out[index] = 0;
  }
}

template <typename Dtype>
void test_delta(const int num, const int channels, const int dim,
    const Dtype threshold_t, const Dtype *in, const Dtype *out) {
  vector<Dtype> delta(channels, 0);
  for (int n = 0; n < num; ++n) {
    for (int c = 0; c < channels; ++c) {
      for (int d = 0; d < dim; ++d) {
        delta[c] += std::abs(in[(n * channels + c) * dim + d]);
      }
    }
  }
  for (int c = 0; c < channels; ++c) {
    delta[c] *= threshold_t;
    if (std::abs(delta[c] - out[c]) > 1e-4) {
      printf("Error: %.10f %.10f\n", delta[c], out[c]);
      CHECK(false);
    }
  }
}

template <typename Dtype>
void TernaryLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  const int count = bottom[0]->count();
  if (!use_global_stats_) {
    Dtype threshold_t = threshold_t_ / Dtype(count / channels_);
    delta_kernel<Dtype><<<channels_, CAFFE_CUDA_NUM_THREADS>>>(num_ * group_,
        channels_, dim_, threshold_t, bottom[0]->gpu_data(),
        delta_.mutable_gpu_data());
    // test_delta<Dtype>(
    //     num_ * group_, channels_, dim_, threshold_t, bottom[0]->cpu_data(),
    //     delta_.cpu_data());
    caffe_gpu_axpby<Dtype>(count, 1. - moving_average_fraction_,
        delta_.gpu_data(), moving_average_fraction_,
        this->blobs_[0]->mutable_gpu_data());
  }
  const Dtype *delta =
      use_global_stats_ ? this->blobs_[0]->gpu_data() : delta_.gpu_data();
  const Dtype *bottom_data = bottom[0]->gpu_data();
  Dtype *top_data          = top[0]->mutable_gpu_data();
  Dtype *beta_data         = top[1]->mutable_gpu_data();
  Dtype *sum_data          = top[2]->mutable_gpu_data();
  const int offset         = channels_ / group_;
  for (int n = 0; n < num_; ++n) {
    for (int g = 0; g < group_; ++g) {
      const int offset2 = (n * group_ + g) * offset * dim_;
      const int offset3 = (n * group_ + g) * dim_;
      forward_kernel<Dtype><<<dim_, CAFFE_CUDA_NUM_THREADS>>>(offset, dim_,
          delta + offset * g, bottom_data + offset2, top_data + offset2,
          beta_data + offset3, sum_data + offset3);
    }
  }
}
template <typename Dtype>
void TernaryLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
    const vector<bool> &propagate_down, const vector<Blob<Dtype> *> &bottom) {
  if (propagate_down[0]) {
    const int count       = bottom[0]->count();
    const Dtype *top_diff = top[0]->gpu_diff();
    caffe_gpu_div<Dtype>(top[1]->count(), top[1]->gpu_data(),
        top[2]->gpu_data(), top[1]->mutable_gpu_data());
    caffe_copy(count, top_diff, bottom[0]->mutable_gpu_diff());
    backward_kernel<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, channels_,
            channels_ / group_, dim_, delta_.gpu_data(), bottom[0]->gpu_data(),
            top[1]->gpu_data(), bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TernaryLayer);
}  // namespace caffe
